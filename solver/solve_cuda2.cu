#include "hip/hip_runtime.h"
/* DGPR: Discontinuous Galerkin Performance Research                          */
/* Copyright (C) 2023  Miles McGruder                                         */
/*                                                                            */
/* This program is free software: you can redistribute it and/or modify       */
/* it under the terms of the GNU General Public License as published by       */
/* the Free Software Foundation, either version 3 of the License, or          */
/* (at your option) any later version.                                        */
/*                                                                            */
/* This program is distributed in the hope that it will be useful,            */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of             */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the              */
/* GNU General Public License for more details.                               */
/*                                                                            */
/* You should have received a copy of the GNU General Public License          */
/* along with this program.  If not, see <https://www.gnu.org/licenses/>.     */

#pragma once

#include <helper_types.cpp>

// naive matrix multiplication
__global__ void cuda_gemm_0(u32 m, u32 k, u32 n, float* A, float* B, float* C)
{
  u32 rC = blockDim.y * blockIdx.y + threadIdx.y;
  u32 cC = blockDim.x * blockIdx.x + threadIdx.x;

  if (rC > m || cC > n)
    return;

  float accC = 0.f;
  for (u32 i = 0; i < k; ++i)
  {
    accC += A[k * rC + i] * B[n * i + cC];
  }

  C[n * rC + cC] = accC;
}

#define TILE_SIZE 16

// reasonable matrix multiplication
__global__ void cuda_gemm_1(u32 m, u32 k, u32 n, float* A, float* B, float* C)
{
  __shared__ float tileA[TILE_SIZE][TILE_SIZE];
  __shared__ float tileB[TILE_SIZE][TILE_SIZE];

  u32 rT = threadIdx.y;
  u32 cT = threadIdx.x;
  u32 rC = TILE_SIZE * blockIdx.y + rT;
  u32 cC = TILE_SIZE * blockIdx.x + cT;

  float accC = 0.;

  for (u32 bk = 0; bk < (k + TILE_SIZE - 1) / TILE_SIZE; ++bk)
  {
    if ((rC) < m && (TILE_SIZE * bk + cT) < k)
      tileA[rT][cT] = A[k * (rC) + (TILE_SIZE * bk + cT)];
    else
      tileA[rT][cT] = 0.;

    if ((TILE_SIZE * bk + rT) < k && (cC) < n)
      tileB[rT][cT] = B[n * (TILE_SIZE * bk + rT) + (cC)];
    else
      tileB[rT][cT] = 0.;

    __syncthreads();

    for (uint i = 0; i < TILE_SIZE; ++i)
      accC += tileA[rT][i] * tileB[i][cT];

    __syncthreads();
  }

  if (rC < m && cC < n)
    C[n * rC + cC] = accC;
}
